#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math_functions.h>
#include <opencv2/core/core.hpp>
#include <opencv2/imgproc.hpp>
#include <opencv2/highgui.hpp>
#include ""

namespace jw {

	double reStart;
	double reEnd;
	double imStart;
	double imEnd;

	int width = 1024;
	int height = 768;
	int maxIter = 80;
	double zoomScale = 3.0;

	int blockSize = 1024;
	int numBlocks = ((width * height) + blockSize - 1) / blockSize;

	int *set;
	unsigned char *data;

	cv::Mat m;


	#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
	inline void gpuAssert(hipError_t code, char *file, int line, bool abort = true)
	{
		if (code != hipSuccess)
		{
			fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
			if (abort) exit(code);
		}
	}

	__global__ void CUDAMandelbrotTest(int *set, unsigned char *data, int maxIter, int width, int height, double reStart, double reEnd, double imStart, double imEnd) {
		
		int t = threadIdx.x;
		int b = blockIdx.x;

		int n = width * height;
		int index = b * 1024 + t;

		if (index >= n) return;

		int x = index % width;
		int y = index / width;

		double c_real = reStart + ((double)x / (double)width)  * (reEnd - reStart);
		double c_imaginary = imStart + ((double)y / (double)height) * (imEnd - imStart);

		double z_real = 0;
		double z_imaginary = 0;
		
		int iter = 0;
		while (sqrt(z_real*z_real + z_imaginary*z_imaginary) <= 2.0 && iter < maxIter) {
			
			double nr = z_real * z_real - z_imaginary * z_imaginary + c_real;
			double ni = 2 * z_real * z_imaginary + c_imaginary;

			z_real = nr;
			z_imaginary = ni;

			iter++;
		}
		set[index] = iter;

		data[index * 3] = (int)(((double)iter / (double)maxIter) * 255); // hue
		data[index * 3+1] = 180; // saturation
		data[index * 3+2] = (iter < maxIter) ? 255 : 0; // value
	}

	void ZoomIn(int x, int y, int width, int height, double scale) {

		double rx, iy;

		rx = (double)x / (double)width;
		iy = (double)y / (double)height;

		double rlen = (reEnd - reStart);
		double ihei = (imEnd - imStart);

		rx = reStart + rx * rlen;
		iy = imStart + iy * ihei;

		reStart = rx - rlen / scale;
		reEnd = rx + rlen / scale;
		imStart = iy - ihei / scale;
		imEnd = iy + ihei / scale;
	}

	void Draw() {

		CUDAMandelbrotTest << <numBlocks, blockSize >> > (set, data, maxIter, width, height, reStart, reEnd, imStart, imEnd);

		hipDeviceSynchronize();

		cv::cvtColor(m, m, cv::COLOR_HSV2BGR);

		cv::imshow("Mandelbrot", m);
	}

	void InitView() {

		double ratio = (double)width / (double)height;

		reStart = -3.5;
		reEnd = 2.5;
		imStart = -((abs(reStart) + abs(reEnd)) * 1.0 / ratio / 2.0);
		imEnd = ((abs(reStart) + abs(reEnd)) * 1.0 / ratio / 2.0);
	}

	void CallBackFunc(int event, int x, int y, int flags, void* userdata)
	{
		if (event == cv::EVENT_LBUTTONDOWN)
		{
			ZoomIn(x, y, width, height, zoomScale);

			Draw();
		}
		else if (event == cv::EVENT_RBUTTONDOWN)
		{
			InitView();

			Draw();
		}/*
		else if (event == cv::EVENT_MBUTTONDOWN)
		{
			//cout << "Middle button of the mouse is clicked - position (" << x << ", " << y << ")" << endl;
		}
		else if (event == cv::EVENT_MOUSEMOVE)
		{
			//cout << "Mouse move over the window - position (" << x << ", " << y << ")" << endl;
		}*/
	}


	void CUDADrawMandelbrot() {

		int N = width * height;

		hipMallocManaged(&set, N * sizeof(int*));
		hipMallocManaged(&data, 3 * N * sizeof(unsigned char*));

		m = cv::Mat(height, width, CV_8UC3, data);

		cv::namedWindow("Mandelbrot", 1);
		cv::setMouseCallback("Mandelbrot", CallBackFunc, NULL);

		InitView();
		Draw();

		cv::waitKey(0);

		m.release();

		hipFree(set);

		gpuErrchk(hipPeekAtLastError());
	}
}

int main(void)
{
	jw::CUDADrawMandelbrot();
	return 0;
}